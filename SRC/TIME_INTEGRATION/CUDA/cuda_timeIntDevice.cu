#include "hip/hip_runtime.h"
/* FastEddy®: SRC/TIME_INTEGRATION/CUDA/cuda_timeIntDevice.cu 
* ©2016 University Corporation for Atmospheric Research
* 
* This file is licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
* http://www.apache.org/licenses/LICENSE-2.0
* 
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/
#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <float.h>
#include <math.h>
#include <fempi.h>
#include <grid.h>
#include <mem_utils.h>
#include <hydro_core.h>
#include <time_integration.h>
#include <fecuda_Device_cu.h>
#include <cuda_gridDevice_cu.h>
#include <cuda_hydroCoreDevice_cu.h>
#include <cuda_timeIntDevice_cu.h>

#include "cuda_RKschemes.cu"

/*#################------------- CUDA_TIME_INTEGRATION module variable definitions ------------------#############*/
/*Parameters*/
__constant__ int timeMethod_d;   // Selector for time integration method. (default: 1= 3rd-order Runge-Kutta )
__constant__ int Nt_d;       // Number of timesteps to perform
__constant__ int NtimeTotVars_d;  // Total Number of prognostic variables to be integrated over time
__constant__ int NtBatch_d;  // Number of timesteps in a batch to perform in a CUDA kernel launch
__constant__ float dt_d;     // timestep resolution in seconds
__constant__ int simTime_itRestart_d;           //Timestep at restart (0 at start) 

/* array fields */
float *timeFlds0_d;   /* Multistage time scheme variable fields 4-D array */
float *timeFrhs0_d;   /* Multistage time scheme variable fields Frhs 4-D array */
float *timeFrhsTmp_d; /* Multistage time scheme variable fields Frhs 4-D array */

/*#################------------- CUDA_TIME_INTEGRATION module function definitions ------------------#############*/
/*----->>>>> int cuda_timeIntDeviceSetup();       ----------------------------------------------------------------
 * Used to hipMalloc and hipMemcpy parameters and coordinate arrays, and for the TIME_INTEGRATION_CUDA module.
*/
extern "C" int cuda_timeIntDeviceSetup(){
   int errorCode = CUDA_TIME_INTEGRATION_SUCCESS;
   int Nelems;
   int NtimeTotVars;
  
   /*Synchronize the Device*/
   gpuErrchk( hipDeviceSynchronize() );
 
   /*Constants*/
   /* timeMethod, total timesteps, batch timesteps, timestep resolution */
   hipMemcpyToSymbol(HIP_SYMBOL(timeMethod_d), &timeMethod, sizeof(int));
   hipMemcpyToSymbol(HIP_SYMBOL(Nt_d), &Nt, sizeof(int));
   hipMemcpyToSymbol(HIP_SYMBOL(NtBatch_d), &NtBatch, sizeof(int));
   hipMemcpyToSymbol(HIP_SYMBOL(dt_d), &dt, sizeof(float));
   hipMemcpyToSymbol(HIP_SYMBOL(simTime_itRestart_d), &simTime_itRestart, sizeof(int));
   gpuErrchk( hipPeekAtLastError() ); /*Check for errors in the cudaMemCpy calls*/

   /*Set the full memory block number of elements for timeInt fields*/
   Nelems = (Nxp+2*Nh)*(Nyp+2*Nh)*(Nzp+2*Nh); 
   /* Allocate the TIME_INTEGRATION arrays */
   /*TIME_INTEGRATION/CUDA internal device arrays*/
   NtimeTotVars = 5 + TKESelector*turbulenceSelector + moistureNvars*moistureSelector + NhydroAuxScalars; 
   fecuda_DeviceMalloc(NtimeTotVars*Nelems*sizeof(float), &timeFlds0_d);
   
   gpuErrchk( hipPeekAtLastError() ); /*Check for errors in the hipMalloc calls*/

   //Ensure secondary time-integration dependent hydro_core parameters get initialized
   errorCode = cuda_hydroCoreDeviceSecondaryStageSetup(dt);
   //Inital Host-to-Device field copies 
   errorCode = cuda_timeIntHydroInitDevice();  //Transfer initial/restart conditions to the device
   //printf("cuda_timeIntDeviceSetup() complete.\n");

   /* Done */
   return(errorCode);
} //end cuda_timeIntDeviceSetup()

/*----->>>>> extern "C" int cuda_timeIntDeviceCleanup();  -----------------------------------------------------------
Used to free all malloced memory by the TIME_INTEGRATION module.
*/
extern "C" int cuda_timeIntDeviceCleanup(){
   int errorCode = TIME_INTEGRATION_SUCCESS;

   /* Free any TIME_INTEGRATION module arrays */
   hipFree(timeFlds0_d); 
   gpuErrchk( hipPeekAtLastError() ); /*Check for errors in the cudaMemCpy calls*/
   return(errorCode);

}//end cuda_timeIntDeviceCleanup()

/*----->>>>> extern "C" int cuda_timeIntDeviceCommence();  -----------------------------------------------------------
* This routine provides the externally callable cuda-kernel call to commence with timeIntegration
*/
extern "C" int cuda_timeIntDeviceCommence(int it){
   int errorCode = TIME_INTEGRATION_SUCCESS;
   int itBatch;
   int RKstage;
#ifdef TIMERS_LEVEL1
   float elapsedTime;
   hipEvent_t startE, stopE
#endif

   /*Synchronize the Device*/
   gpuErrchk( hipDeviceSynchronize() );

   for(itBatch=0; itBatch < NtBatch; itBatch++){     //Batch timestepping loop
     if((lsfSelector == 1) && (lsf_horMnSubTerms == 1) && (simTime_it > simTime_itRestart) && (simTime_it%(int)roundf(lsf_freq/dt)==0)){
       errorCode = cuda_lsfSlabMeans();
     }
     gpuErrchk( hipDeviceSynchronize() );
     /*Execute the timeMethod kernel of choice on the GPU*/
     if(timeMethod == 0){    /*******  Issue the  3rd-order Runge-Kutta WS2002 **************/
       if(cellpertSelector==1 && simTime_it%cellpert_nts==0){ /***** Issue cell perturbation method here *****/
         errorCode = cuda_hydroCoreDeviceBuildCPmethod(simTime_it); // call to buildCPmethod
       }
       for(RKstage=0; RKstage < 3; RKstage++){
          /*Build the right hand side forcing*/
          errorCode = cuda_hydroCoreDeviceBuildFrhs(simTime,simTime_it,simTime_itRestart,dt,RKstage,numRKstages);
          /*Perform the time integration*/
#ifdef TIMERS_LEVEL1
          createAndStartEvent(&startE, &stopE);
#endif
          cudaDevice_timeIntegrationCommenceRK3_WS2002<<<grid, tBlock>>>(Nhydro, hydroFlds_d, hydroFldsFrhs_d,
                                                TKESelector*turbulenceSelector, sgstkeScalars_d, sgstkeScalarsFrhs_d,
                                                moistureNvars*moistureSelector, moistScalars_d, moistScalarsFrhs_d,
						NhydroAuxScalars, hydroAuxScalars_d, hydroAuxScalarsFrhs_d,
                                                timeFlds0_d, RKstage);
          gpuErrchk( hipGetLastError() );
#ifdef TIMERS_LEVEL1
          stopSynchReportDestroyEvent(&startE, &stopE, &elapsedTime);
          printf("cuda_timeIntCommenceRK3_WS2002()  Kernel execution time (ms): %12.8f\n", elapsedTime);
#endif
       } //end for RKstage 
     } //end if(timeMethod == 0){...
     simTime = simTime + dt;   //Increment the master simulation time
     simTime_it = simTime_it + 1;   //Increment the master simulation time step
   }//end for itBatch...

   //Retrieve desired HYDRO_CORE fields from device
   errorCode = cuda_timeIntHydroSynchFromDevice();
   
   return(errorCode);
}//end cuda_timeIntDeviceCommence()

/*----->>>>> extern "C" int cuda_timeIntHydroInitDevice();  -----------------------------------------------------------
* This function handles the one-time initializations of on-device (GPU) memory by executing the appropriate sequence 
* of hipMemcpyHostToDevice data transfers.
*/
extern "C" int cuda_timeIntHydroInitDevice(){
   int errorCode = TIME_INTEGRATION_SUCCESS;
   int Nelems;
   int Nelems2d;
   /*Set the full memory block number of elements for transfers of 2-d and 3-d fields*/
   Nelems = (Nxp+2*Nh)*(Nyp+2*Nh)*(Nzp+2*Nh);
   Nelems2d = (Nxp+2*Nh)*(Nyp+2*Nh);
   /*Copy the host hydroFlds to the device */
   hipMemcpy(hydroFlds_d, hydroFlds, Nelems*Nhydro*sizeof(float), hipMemcpyHostToDevice);
   if(TKESelector > 0){ /*Copy any required SGS TKE equation fields to device */ 
     hipMemcpy(sgstkeScalars_d, sgstkeScalars, Nelems*TKESelector*sizeof(float), hipMemcpyHostToDevice);
   }
   if(moistureSelector > 0){ /*Copy any required moisture fields to device */ 
     hipMemcpy(moistScalars_d, moistScalars, Nelems*moistureNvars*sizeof(float), hipMemcpyHostToDevice);
   }
   if(surflayerSelector > 0){ /*Copy any required host auxiliary sclar fields to the device */
     hipMemcpy(tskin_d, tskin, Nelems2d*sizeof(float), hipMemcpyHostToDevice);
     hipMemcpy(fricVel_d, fricVel, Nelems2d*sizeof(float), hipMemcpyHostToDevice);
     hipMemcpy(htFlux_d, htFlux, Nelems2d*sizeof(float), hipMemcpyHostToDevice);
     hipMemcpy(z0m_d, z0m, Nelems2d*sizeof(float), hipMemcpyHostToDevice);
     hipMemcpy(z0t_d, z0t, Nelems2d*sizeof(float), hipMemcpyHostToDevice);
     if (moistureSelector > 0){
       hipMemcpy(qskin_d, qskin, Nelems2d*sizeof(float), hipMemcpyHostToDevice);
       hipMemcpy(qFlux_d, qFlux, Nelems2d*sizeof(float), hipMemcpyHostToDevice);
     }
   }// end if surflayerSelector > 0
   if(NhydroAuxScalars > 0){ /*Copy any required host auxiliary sclar fields to the device */
     hipMemcpy(hydroAuxScalars_d, hydroAuxScalars, Nelems*NhydroAuxScalars*sizeof(float), hipMemcpyHostToDevice);
   }// end if hydroAuxScalars > 0
   gpuErrchk( hipPeekAtLastError() ); /*Check for errors in the cudaMemCpy calls*/
   gpuErrchk( hipDeviceSynchronize() );
   return(errorCode);
}//end cuda_timeIntHydroInitDevice()

/*----->>>>> extern "C" int cuda_timeIntHydroSynchFromDevice();  --------------------------------------------------
* This function handles the synchronization to host of on-device (GPU) fields  by executing the appropriate sequence 
* of cudaMemcpyDeviceiToHost data transfers.
*/
extern "C" int cuda_timeIntHydroSynchFromDevice(){
   int errorCode = TIME_INTEGRATION_SUCCESS;
   int Nelems;
   int Nelems2d;

   /*Set the full memory block number of elements for transfers of 2-d and 3-d fields*/
   Nelems = (Nxp+2*Nh)*(Nyp+2*Nh)*(Nzp+2*Nh);
   Nelems2d = (Nxp+2*Nh)*(Nyp+2*Nh);

   /* Send any desired GPU-computed HYDRO_CORE arrays from Device up to Host*/
   gpuErrchk( hipMemcpy(hydroPres, hydroPres_d, Nelems*sizeof(float), hipMemcpyDeviceToHost) );
   gpuErrchk( hipMemcpy(hydroFlds, hydroFlds_d, Nelems*Nhydro*sizeof(float), hipMemcpyDeviceToHost) );
   if((hydroForcingWrite==1)||(hydroForcingLog==1)){
     gpuErrchk( hipMemcpy(hydroFldsFrhs, hydroFldsFrhs_d, Nelems*Nhydro*sizeof(float), hipMemcpyDeviceToHost) );
   } //endif we need to send up the Frhs
   if (TKESelector > 0){ 
     gpuErrchk( hipMemcpy(sgstkeScalars, sgstkeScalars_d, Nelems*TKESelector*sizeof(float), hipMemcpyDeviceToHost) );
     if ((hydroForcingWrite==1)||(hydroForcingLog==1)){
       gpuErrchk( hipMemcpy(sgstkeScalarsFrhs, sgstkeScalarsFrhs_d, Nelems*TKESelector*sizeof(float), hipMemcpyDeviceToHost) );
     }
   }
   if (moistureSelector > 0){ 
     gpuErrchk( hipMemcpy(moistScalars, moistScalars_d, Nelems*moistureNvars*sizeof(float), hipMemcpyDeviceToHost) );
     if ((hydroForcingWrite==1)||(hydroForcingLog==1)){
       gpuErrchk( hipMemcpy(moistScalarsFrhs, moistScalarsFrhs_d, Nelems*moistureNvars*sizeof(float), hipMemcpyDeviceToHost) );
     }
   }
   if(surflayerSelector > 0){
     gpuErrchk( hipMemcpy(fricVel, fricVel_d, Nelems2d*sizeof(float), hipMemcpyDeviceToHost) );
     gpuErrchk( hipMemcpy(htFlux, htFlux_d, Nelems2d*sizeof(float), hipMemcpyDeviceToHost) );
     gpuErrchk( hipMemcpy(tskin, tskin_d, Nelems2d*sizeof(float), hipMemcpyDeviceToHost) );
     gpuErrchk( hipMemcpy(invOblen, invOblen_d, Nelems2d*sizeof(float), hipMemcpyDeviceToHost) );
     gpuErrchk( hipMemcpy(z0m, z0m_d, Nelems2d*sizeof(float), hipMemcpyDeviceToHost) );
     gpuErrchk( hipMemcpy(z0t, z0t_d, Nelems2d*sizeof(float), hipMemcpyDeviceToHost) );
     if (moistureSelector > 0){
       gpuErrchk( hipMemcpy(qFlux, qFlux_d, Nelems2d*sizeof(float), hipMemcpyDeviceToHost) );
       gpuErrchk( hipMemcpy(qskin, qskin_d, Nelems2d*sizeof(float), hipMemcpyDeviceToHost) );
     }
   }//endif surflayerSelector > 0
   if(NhydroAuxScalars > 0){
     gpuErrchk( hipMemcpy(hydroAuxScalars, hydroAuxScalars_d, Nelems*NhydroAuxScalars*sizeof(float), hipMemcpyDeviceToHost) );
     if((hydroForcingWrite==1)||(hydroForcingLog==1)){
       gpuErrchk( hipMemcpy(hydroAuxScalarsFrhs, hydroAuxScalarsFrhs_d, Nelems*NhydroAuxScalars*sizeof(float), hipMemcpyDeviceToHost) );
     } //endif we need to send up the Frhs
   } //end if NhydroAuxScalars > 0
   if(hydroSubGridWrite==1){
     if(turbulenceSelector > 0){
       // The 6 Tau_i-j and 3 Tau_TH,j fields
       gpuErrchk( hipMemcpy(hydroTauFlds, hydroTauFlds_d, Nelems*9*sizeof(float), hipMemcpyDeviceToHost) );
     }//endif 
     if(moistureSGSturb==1){
       // The moistureNvars*3 tau moisture fields (3 spatial components per moist species)
       gpuErrchk( hipMemcpy(moistTauFlds, moistTauFlds_d, Nelems*moistureNvars*3*sizeof(float), hipMemcpyDeviceToHost) );
     }
   } //endif hydroSubGridWrite==1 
   gpuErrchk( hipPeekAtLastError() ); /*Check for errors in the cudaMemCpy calls*/
//#ifdef DEBUG
#if 1
   MPI_Barrier(MPI_COMM_WORLD);
   printf("Rank %d/%d: Batch complete results sent via hipMemcpyDeviceToHost.\n",mpi_rank_world, mpi_size_world);
   fflush(stdout);
   MPI_Barrier(MPI_COMM_WORLD);
#endif

   return(errorCode);
}//end cuda_timeIntHydrosynchFromDevice()
